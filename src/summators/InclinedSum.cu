#include "hip/hip_runtime.h"
#include "src/summators/InclinedSum.cuh"
#include <cmath>

template <class T>
InclinedSum<T>::InclinedSum(const Parameters<T>* _props, const Well<T>* _well) : BaseSum<T>(_props, _well)
{
}

template <class T>
InclinedSum<T>::~InclinedSum()
{
}

template <class T>
void InclinedSum<T>::prepare()
{
	const int threadsPerBlock = 128;
	const int blocksPerGrid = 16 * size;

	// Allocate memory on device
	cudaErrorsChecker(hipMalloc((void**)&F2d_dev, sizeof(T) * size));
	cudaErrorsChecker(hipMalloc((void**)&F2d_buf, sizeof(T) * blocksPerGrid));
	cudaErrorsChecker(hipMalloc((void**)&F3d_dev, sizeof(T) * size));
	cudaErrorsChecker(hipMalloc((void**)&F3d_buf, sizeof(T) * blocksPerGrid));
	cudaErrorsChecker(hipMalloc((void**)&segs, sizeof(WellSegment<T>) * props->K));
	
	cudaErrorsChecker(hipMemcpy(segs, well->segs, sizeof(WellSegment<T>) * props->K, hipMemcpyHostToDevice));

	// Get device properties
	hipDeviceProp_t deviceProp;
	cudaErrorsChecker(hipGetDeviceProperties(&deviceProp, 0));

	dim3 blockSize(threadsPerBlock, 1, 1);			dim3 blockSizeR(1, 1, 1);
	dim3 gridSize(size, blocksPerGrid / size, 1);	dim3 gridSizeR(size, 1, 1);
	int sharedMem = threadsPerBlock * sizeof(T) + props->K * sizeof(WellSegment<T>);

	// Perform calculations
	prep2D<T, threadsPerBlock><<<gridSize, blockSize, sharedMem>>>(F2d_buf, *props, segs);
	prep3D<T, threadsPerBlock><<<gridSize, blockSize, sharedMem>>>(F3d_buf, *props, segs);

	hipDeviceSynchronize();

	reduce<T><<<gridSizeR, blockSizeR>>>(F2d_buf, F2d_dev, blocksPerGrid / size);
	reduce<T><<<gridSizeR, blockSizeR>>>(F3d_buf, F3d_dev, blocksPerGrid / size);

	cudaErrorsChecker(hipGetLastError());
	hipDeviceSynchronize();

	// Transfer results on host memory
	cudaErrorsChecker(hipMemcpy(F2d, F2d_dev, sizeof(T) * size, hipMemcpyDeviceToHost));
	cudaErrorsChecker(hipMemcpy(F3d, F3d_dev, sizeof(T) * size, hipMemcpyDeviceToHost));

	// Free device memory
	cudaErrorsChecker(hipFree(F2d_dev));
	cudaErrorsChecker(hipFree(F2d_buf));
	cudaErrorsChecker(hipFree(F3d_dev));
	cudaErrorsChecker(hipFree(F3d_buf));
	cudaErrorsChecker(hipFree(segs));
}

template <class T>
T InclinedSum<T>::get2D(int seg_idx)
{
	T sum = 0.0;

	for (int k = 0; k < props->K; k++)
	{
		const WellSegment<T>& seg = well->segs[k];
		sum += F2d[seg_idx * props->K + k] * seg.rate / seg.length;
	}

	sum *= (props->visc * props->sizes.x / HIP_PI / HIP_PI / props->sizes.z / props->kx / sin(props->alpha));

	return sum;
}

template <class T>
T InclinedSum<T>::get3D(int seg_idx)
{
	T sum = 0.0;

	for (int k = 0; k < props->K; k++)
	{
		const WellSegment<T>& seg = well->segs[k];
		sum += F3d[seg_idx * props->K + k] * seg.rate / seg.length;
	}

	sum *= (2.0 * props->visc / HIP_PI / props->sizes.x / props->sizes.z / props->kx / cos(props->alpha));

	return sum;
}

template class InclinedSum<float>;
template class InclinedSum<double>;
